#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <iostream>
#include <stdio.h>
#include <time.h>
#include <math.h>
#include <vector>
#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"// read pictures stb library
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"

#define BLOCKDIM_X 32
#define BLOCKDIM_Y 32
#define BLOCKDIM BLOCKDIM_X * BLOCKDIM_Y
#define W 64
#define H 64
#define N W * H
#define CHANNEL_NUM 1
using namespace std;

void insertionSort_(int *arr, int n);
void swap_(int *a, int *b);
uint8_t* readImage(char* file, int &width, int &height, int bpp);
void writeImage(char* file, int width, int height, uint8_t *image);
void compareVectors(int *a, int *b);
void generateRandom(int *a,int rows, int cols);
void serial_median_filter3x3();
void parallel_median_filter3x3();
void printMAT(int *a);
void printVEC(int *a,int w);
void copiarValores(int *a, int *b);

int *h_img, *filtered_img_serial, *filtered_img_par;
int *d_img, *d_img_res;
int size = W*H*sizeof(int);

// global timers
double serialTimer = 0.0;
float parallelTimer = 0.0;

#define CUDA_CALL(x) {hipError_t cuda_error__ = (x); if (cuda_error__) printf("CUDA error: " #x " returned \"%s\"\n", hipGetErrorString(cuda_error__));}

#define KERNEL_R 3

__device__ void swap(int *a, int *b){
    int d = *a;
    *a = *b;
    *b = d;
}

__device__ void insertionSort(int *arr, int n){
    for(int i = 1; i < n; i++){
        int j = i - 1;
        int key = arr[i];
        while(j >= 0 && arr[j] > key){
            if(arr[j] > arr[j + 1])
                swap(&arr[j], &arr[j + 1]);
            j--;
        }
    }
}

__device__ void sort(int *a, int *b, int *c) {
    int d;
    if(*a > *b){
        d = *a;
        *a = *b;
        *b  = d;
    }
    if(*a > *c){
        d = *a;
        *a = *c;
        *c  = d;
    }
    if(*b > *c){
        d = *b;
        *b = *c;
        *c  = d;
    }
}


__global__ void medianFilter3x3(const int *src, int w, int h, int *dst){
    const int r = KERNEL_R;
    int imgBlock[r * r];
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;
    int rHalf = r / 2;
    int i, j, k, l;
    if(x - rHalf > 0 && x + rHalf < w && y - rHalf > 0 && y + rHalf < h){
        for (i = x - rHalf, k = 0; i <= x + rHalf; i++, k++) {
            for (j = y - rHalf, l = 0; j <= y + rHalf; j++, l++) {
                imgBlock[k*r + l] = src[i* w + j];
            }
        }

        insertionSort(imgBlock,r*r);

        //Columns
        //for(int row = 0; row < r; row++){
        //    sort(&imgBlock[row * r], &imgBlock[row * r + 1], &imgBlock[row * r + 2]);
        //}
        //Rows
        //for(int col = 0; col < r; col++){
         //   sort(&imgBlock[col], &imgBlock[col + r], &imgBlock[col + r * 2]);
        //}
        //Diagonal
        //sort(&imgBlock[0], &imgBlock[1 + r], &imgBlock[2 + 2 * r]);

        //Set median
        dst[x* w + y ] = imgBlock[rHalf + rHalf * r];
    }
    else if(x < w && y < h){
        dst[x*w + y ] = src[x*w + y];
    }
}

int main() {

    // Reservar memoria en host
    h_img = (int *) malloc(size);
    filtered_img_par = (int *) malloc(size);
    filtered_img_serial = (int *) malloc(size);

    //generar img aleatoria
    generateRandom(h_img,W,H);

    // Reservar memoria en device
    hipMalloc((void **)&d_img, size);
    // Transferir datos de host a device
    hipMemcpy(d_img, h_img, size, hipMemcpyHostToDevice);

    serial_median_filter3x3();
    parallel_median_filter3x3();
    compareVectors(filtered_img_par,filtered_img_serial);
    cout << "Serial: " << serialTimer << " Parallel: " << parallelTimer / 1000 <<endl;
    cout << "Speed-up: " << serialTimer / (parallelTimer /1000)<< "X"<<endl;
    cout << "\n"<<endl;

    CUDA_CALL(hipFree(d_img));
    CUDA_CALL(hipFree(d_img_res));

    free(h_img);
    free(filtered_img_par);
    free(filtered_img_serial);

    return 0;
}

void parallel_median_filter3x3(){
    // Reservar memoria en device
    hipMalloc((void **)&d_img_res, size);
    dim3 blockSize = dim3(BLOCKDIM_X, BLOCKDIM_Y);
    dim3 gridSize = dim3(ceil(W / BLOCKDIM_X)+ 1, ceil(H / BLOCKDIM_Y)+ 1);

    // Definir timers
    hipEvent_t start, stop;

    // Eventos para tomar tiempo
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start,0);
    medianFilter3x3<<<gridSize, blockSize>>>(d_img, W, H, d_img_res);
    hipEventRecord(stop,0);

    hipEventSynchronize(stop);

    hipEventElapsedTime(&parallelTimer, start, stop);

    cout<< "Tiempo en generar el filtro en paralelo: " << parallelTimer << " ms, " << parallelTimer / 1000 << " secs" <<endl;

    CUDA_CALL(hipMemcpy(filtered_img_par, d_img_res, size, hipMemcpyDeviceToHost));

    //printf("Source image: \n");
    //printMAT(h_img);

    //printf("Blurred image par: \n");
    //printMAT(filtered_img_par);
}


void serial_median_filter3x3(){
    clock_t start = clock();
    for(int i=0;i<W;i++){
        for(int j=0;j<H;j++){
            int box3x3[9];
            if(i>0 && j>0 && i<W-1 && j<H-1){
                box3x3[0] = h_img[(i-1)*W+(j-1)];
                box3x3[1] = h_img[(i-1)*W+(j)];
                box3x3[2] = h_img[(i-1)*W+(j+1)];
                box3x3[3] = h_img[(i)*W+(j-1)];
                box3x3[4] = h_img[(i)*W+(j)];
                box3x3[5] = h_img[(i)*W+(j+1)];
                box3x3[6] = h_img[(i+1)*W+(j-1)];
                box3x3[7] = h_img[(i+1)*W+(j)];
                box3x3[8] = h_img[(i+1)*W+(j+1)];

                insertionSort_(box3x3,9);
                int median = box3x3[4];
                filtered_img_serial[i*W+j]=median;
            }
            else{
                filtered_img_serial[i*W+j]=h_img[i*W+j];
            }

        }
    }
    clock_t end = clock();
    serialTimer = double (end-start) / double(CLOCKS_PER_SEC);
    cout << "Tiempo en obtener filtro img serial: " << serialTimer << endl;
    //printf("Blurred image ser: \n");
    //printMAT(filtered_img_serial);
}


void swap_(int *a, int *b){
    int d = *a;
    *a = *b;
    *b = d;
}

void insertionSort_(int *arr, int n){
    for(int i = 1; i < n; i++){
        int j = i - 1;
        int key = arr[i];
        while(j >= 0 && arr[j] > key){
            if(arr[j] > arr[j + 1])
                swap_(&arr[j], &arr[j + 1]);
            j--;
        }
    }
}

uint8_t* readImage(char *file, int &width, int &height, int bpp){
    //
    uint8_t *rgb_image = stbi_load(file, &width, &height, &bpp, CHANNEL_NUM);
    cout<< "Image size: " << width << " x " << height  << " = " << width * height  << " pixels"<< endl;
    return rgb_image;
}

void writeImage(char* file, int width, int height,  uint8_t *image){
    stbi_write_png(file, width, height, CHANNEL_NUM, image, width*CHANNEL_NUM);
}

void generateRandom(int *a,int rows, int cols){
    // Initialize seed
    srand(time(NULL));
    for(int i=0; i<rows*cols; i++){
        a[i] = rand() % 256;
    }
}
void compareVectors(int *a, int *b){
    cout<<"Total elements "<<W*H<< endl;
    int diff = 0;
    for(int i= 0; i<W*H; i++)
        if(a[i] != b[i]){
            diff++;
        }

    if(diff>0){
        cout<< diff <<" elements different" << endl;
    }
    else
        cout << "Vectors are equal!..." << endl;
}

void printMAT(int *a){
    cout<<"[\n"<<endl;
    for(int i = 0; i < W; i++){
        cout<<"["<<"";
        for(int j = 0; j < H; j++){
            cout<<a[i* W + j]<<" ";
        }
        cout<<"]\n"<<endl;
    }
    cout<<"]\n"<<endl;
}
void printVEC(int *a,int w){
    cout<<"[\n"<<endl;
    for(int i = 0; i < w; i++){
        cout<<a[i]<<" ";
    }
    cout<<"]\n"<<endl;
}

void copiarValores(int *a, int *b){
    for(int i=0;i<W*H;i++){
        b[i]=a[i];
    }
}