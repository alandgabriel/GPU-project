#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <iostream>
#include <stdio.h>
#include <time.h>
#include <math.h>
#include <vector>
#include "main.h"
//#include "OpenCV.h"
//#include "OpenCV.cpp"
#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"// read pictures stb library
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h" // write prictures stb library

#define BLOCKDIM_X 32
#define BLOCKDIM_Y 32
//#define W 4096
//#define H 4096
#define FILTER 3
#define CHANNEL_NUM 1

using namespace std;

void insertionSort_(uint8_t *arr, int n);
void swap_(uint8_t *a, uint8_t *b);
void compareVectors(uint8_t *a, uint8_t *b);
void generateRandom(uint8_t *a,int rows, int cols);
void serial_median_filter3x3();
void parallel_median_filter3x3();
void printMAT(uint8_t *a);
void printVEC(uint8_t *a,int w);
void copiarValores(uint8_t *a, uint8_t *b);
//Leer y escribir imagenes
uint8_t* readImage(char* file, int &width, int &height, int bpp);
void writeImage(char* file, int width, int height, uint8_t *image);

//Cargar imagen
//OpenCV cv2("/tmp/tmp.UfSt4NNo2q/input/lena.jpg");
//int H = cv2.getRows();
//int W = cv2.getCols();

int W, H, bpp;
char *inFile= "/tmp/tmp.UfSt4NNo2q/input/lena_noi.jpg"; //
char *outImage= "/tmp/tmp.UfSt4NNo2q/outputs/filtro_lena_noi.jpg"; //


// Leer una imagen
uint8_t * image = readImage(inFile, W, H, bpp);

//generar imagen aleatoria
//uint8_t image[W*H];
//generateRandom(image,W,H);

//Variables globales
uint8_t *h_img, *filtered_img_serial, *filtered_img_par;
uint8_t *d_img, *d_img_res;
int size = W*H*sizeof(int);

// Timers globales
double serialTimer = 0.0;
float parallelTimer = 0.0;

#define CUDA_CALL(x) {hipError_t cuda_error__ = (x); if (cuda_error__) printf("CUDA error: " #x " returned \"%s\"\n", hipGetErrorString(cuda_error__));}

#define KERNEL_R 3

__device__ void swap(uint8_t *a, uint8_t *b){
    int d = *a;
    *a = *b;
    *b = d;
}

__device__ void insertionSort(uint8_t *arr, int n){
    for(int i = 1; i < n; i++){
        int j = i - 1;
        int key = arr[i];
        while(j >= 0 && arr[j] > key){
            if(arr[j] > arr[j + 1])
                swap(&arr[j], &arr[j + 1]);
            j--;
        }
    }
}

__device__ void sort(uint8_t *a, uint8_t *b, uint8_t *c) {
    int d;
    if(*a > *b){
        d = *a;
        *a = *b;
        *b  = d;
    }
    if(*a > *c){
        d = *a;
        *a = *c;
        *c  = d;
    }
    if(*b > *c){
        d = *b;
        *b = *c;
        *c  = d;
    }
}


__global__ void medianFilter3x3(const uint8_t *src, int w, int h, uint8_t *dst){
    const int r = KERNEL_R;
    int rHalf = r / 2;
    uint8_t imgBlock[r * r];
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;
    int i, j, k, l;
    if(x  > 0 && x  < w-1 && y  > 0 && y < h-1){
        for (i = x - rHalf, k = 0; i <= x + rHalf; i++, k++) {
            for (j = y - rHalf, l = 0; j <= y + rHalf; j++, l++) {
                imgBlock[k* r + l ] = src[i* w + j ];
            }
        }

        insertionSort(imgBlock,r*r);

        dst[x* w + y ] = imgBlock[rHalf + rHalf * r];
    }
    else if(x < w && y < h){
        dst[x*w + y ] = src[x*w + y];
    }
}

int main() {

    // Reservar memoria en host
    h_img = (uint8_t *) malloc(size);
    filtered_img_par = (uint8_t *) malloc(size);
    filtered_img_serial = (uint8_t *) malloc(size);

    //Convertir imagen a arreglo
    //cv2.getImage(h_img);

    //Copiar imagen
    copiarValores(image,h_img);

    //printf("Source image: \n");
    //printMAT(h_img);

    // Reservar memoria en device
    hipMalloc((void **)&d_img, size);
    // Transferir datos de host a device
    hipMemcpy(d_img, h_img, size, hipMemcpyHostToDevice);

    serial_median_filter3x3();
    parallel_median_filter3x3();
    compareVectors(filtered_img_par,filtered_img_serial);
    cout << "Serial: " << serialTimer << " Parallel: " << parallelTimer / 1000 <<endl;
    cout << "Speed-up: " << serialTimer / (parallelTimer /1000)<< "X"<<endl;
    cout << "\n"<<endl;

    CUDA_CALL(hipFree(d_img));
    CUDA_CALL(hipFree(d_img_res));

    //Guardar resultado
    //cv2.saveToimg(filtered_img_serial,"/tmp/tmp.UfSt4NNo2q/filtro.jpg",W,H);

    // Guardar imagen en escala de grises
    writeImage(outImage, W, H, filtered_img_serial);

    free(h_img);
    free(filtered_img_par);
    free(filtered_img_serial);

    return 0;
}

void parallel_median_filter3x3(){
    // Reservar memoria en device
    hipMalloc((void **)&d_img_res, size);
    dim3 blockSize = dim3(BLOCKDIM_X, BLOCKDIM_Y);
    dim3 gridSize = dim3(ceil(W / BLOCKDIM_X)+ 1, ceil(H / BLOCKDIM_Y)+ 1);

    // Definir timers
    hipEvent_t start, stop;

    // Eventos para tomar tiempo
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start,0);
    medianFilter3x3<<<gridSize, blockSize>>>(d_img, W, H, d_img_res);
    hipEventRecord(stop,0);

    hipEventSynchronize(stop);

    hipEventElapsedTime(&parallelTimer, start, stop);

    cout<< "Tiempo en generar el filtro en paralelo: " << parallelTimer << " ms, " << parallelTimer / 1000 << " secs" <<endl;

    CUDA_CALL(hipMemcpy(filtered_img_par, d_img_res, size, hipMemcpyDeviceToHost));

    //printf("Source image: \n");
    //printMAT(h_img);

    //printf("Blurred image par: \n");
    //printMAT(filtered_img_par);
}


void serial_median_filter3x3(){
    clock_t start = clock();
    int half = FILTER/2;
    for(int i=0 ;i<W;i++){
        for(int j=0;j<H;j++){
            uint8_t box3x3[FILTER*FILTER];
            if(i>0 && j>0 && i<W-1 && j<H-1){
                box3x3[0] = h_img[(i-1)*W+(j-1)];
                box3x3[1] = h_img[(i-1)*W+(j)];
                box3x3[2] = h_img[(i-1)*W+(j+1)];
                box3x3[3] = h_img[(i)*W+(j-1)];
                box3x3[4] = h_img[(i)*W+(j)];
                box3x3[5] = h_img[(i)*W+(j+1)];
                box3x3[6] = h_img[(i+1)*W+(j-1)];
                box3x3[7] = h_img[(i+1)*W+(j)];
                box3x3[8] = h_img[(i+1)*W+(j+1)];
                insertionSort_(box3x3,FILTER*FILTER);
                uint8_t median = box3x3[FILTER+half];
                filtered_img_serial[i*W+j]=median;
            }
            else{
                filtered_img_serial[i*W+j]=h_img[i*W+j];
            }
        }
    }
    clock_t end = clock();
    serialTimer = double (end-start) / double(CLOCKS_PER_SEC);
    cout << "Tiempo en obtener filtro img serial: " << serialTimer << endl;
    //printf("Blurred image ser: \n");
    //printMAT(filtered_img_serial);
}


void swap_(uint8_t *a, uint8_t *b){
    int d = *a;
    *a = *b;
    *b = d;
}

void insertionSort_(uint8_t *arr, int n){
    for(int i = 1; i < n; i++){
        int j = i - 1;
        int key = arr[i];
        while(j >= 0 && arr[j] > key){
            if(arr[j] > arr[j + 1])
                swap_(&arr[j], &arr[j + 1]);
            j--;
        }
    }
}

void generateRandom(uint8_t *a,int rows, int cols){
    // Initialize seed
    srand(time(NULL));
    for(int i=0; i<rows*cols; i++){
        a[i] = (uint8_t) (rand() % 256);
    }
}
void compareVectors(uint8_t *a, uint8_t *b){
    cout<<"Total elements "<<W*H<< endl;
    int diff = 0;
    for(int i= 0; i<W*H; i++)
        if(a[i] != b[i]){
            diff++;
        }

    if(diff>0){
        cout<< diff <<" elements different" << endl;
    }
    else
        cout << "Vectors are equal!..." << endl;
}

void printMAT(uint8_t *a){
    cout<<"["<<"";
    for(int i = 0; i < W; i++){
        cout<<"["<<"";
        for(int j = 0; j < H; j++){
            if((i* W + j)<(i+1)*W-1){
                cout<<a[i* W + j]<<", ";
            }
            else{
                cout<<a[i* W + j]<<"";
            }
        }
        if (i<W-1){
            cout<<"],"<<endl;
        }
        else{
            cout<<"]"<<"";
        }

    }
    cout<<"]\n"<<endl;
}
void printVEC(uint8_t *a,int w){
    cout<<"["<<endl;
    for(int i = 0; i < w; i++){
        cout<<a[i]<<", ";
    }
    cout<<"]\n"<<endl;
}

void copiarValores(uint8_t *a, uint8_t *b){
    for(int i=0;i<W*H;i++){
        b[i]=a[i];
    }
}

uint8_t* readImage(char *file, int &width, int &height, int bpp){
    //
    uint8_t *rgb_image = stbi_load(file, &width, &height, &bpp, CHANNEL_NUM);
    cout<< "Image size: " << width << " x " << height  << " = " << width * height  << " pixels"<< endl;
    return rgb_image;
}

void writeImage(char* file, int width, int height,  uint8_t *image){
    stbi_write_png(file, width, height, CHANNEL_NUM, image, width*CHANNEL_NUM);

}